#include "hip/hip_runtime.h"
/**
 * @file validation_cuda.c
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include "sha256_cuda.cu"

/** Cuda kernals for executing functions (from host cpu) on device (gpu) */
__global__ void kernel(unsigned char *d, int d_len, unsigned char *out) {
  SHA256(d, d_len, out);
}

// __global__ void kernel(unsigned char digest[], FILE *fp_new) {
//   validate(digest, fp_new);
// }

/** Validate block */
void validate(unsigned char digest[], FILE *fp_new) {
    /** Correct hash to test against */
    unsigned char answer[(SHA256_DIGEST_LENGTH * 2) + 1];

    fgets((char *)answer, (SHA256_DIGEST_LENGTH * 2) + 2, fp_new);
    printf("Correct Hash: %s\n", answer);

    char res[(SHA256_DIGEST_LENGTH * 2) + 1] = "";
    for (int i = 0; i < SHA256_DIGEST_LENGTH; ++i) {
        sprintf(res + i * 2, "%02x", digest[i]);
    }

    if (strcmp((char *)res, (char *)answer) != 0) {
         printf("Block Validated!\n");
    }
    else {
        printf("Block Incorrect!\n");
    }
}

/** SHA256 hash function */
unsigned char *sha256(unsigned char *data, int d_len, FILE *fp_new, int checksum) {
  /** CUDA kernels are asynchronous -- but GPU-related tasks placed in one stream
  are executed sequentially. hipDeviceSynchronize() synchronizes the executions */
  hipDeviceSynchronize();
  unsigned char *d_c;

  /** Allocates size bytes of managed memory on the GPU device */
  hipMallocManaged((void **)&d_c, d_len);

  /** Copy memory buffers */
  hipMemcpy(d_c, data, d_len, hipMemcpyHostToDevice);

  unsigned char *digest_c;
  hipMallocManaged((void **)&digest_c, SHA256_DIGEST_LENGTH);
  unsigned char digest[SHA256_DIGEST_LENGTH] = {};

  /** Call kernel to run SHA256 function */
  kernel<<<1, 1>>>(d_c, d_len, digest_c);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  /** Copy results back from GPU device */
  hipMemcpy(digest, digest_c, SHA256_DIGEST_LENGTH, hipMemcpyDeviceToHost);
    
  /** Print hash result */
  for (int i = 0; i < SHA256_DIGEST_LENGTH; ++i) {
      printf("%02x", digest[i]);
  }
  printf("\n");

  /** Free memory allocation */
  hipFree(d_c);
  hipFree(digest_c);
  
  return digest;
}

int main () {
    /** Declare buffers */
    unsigned char buffer[256];
    unsigned char header[256];
    unsigned char nonce[256];
    unsigned char concat[256];
    
    /** Open files for reading */
    const char filename[] = "../data/blockchain_partial.csv";
    FILE *fp = fopen(filename, "r");

    const char filename_newer[] = "../data/blockchain_full.csv";
    FILE *fp_newer = fopen(filename_newer, "r");

    const char filename_new[] = "../data/block_hash.csv";
    FILE *fp_new = fopen(filename_new, "r+");

    int counter = 0;
    clock_t t;
    t = clock();
    while (fgets((char *)buffer, 256, fp)) {
        buffer[strcspn((char *)buffer, "\n")] = 0;
        printf("--------------------------------------------------------------------------------\n");
        printf("Information for Block: %d\n\n", counter);

        printf("BLOCK CONTENTS......... \n");
        printf("Block: %s\n", buffer);

        /** Copy buffer read into header and nonce */
        memcpy(header, buffer, sizeof(buffer));
        memcpy(nonce, buffer, sizeof(buffer));

        /** Hash of header and nonce */
        printf("\nHASHING HEADER + NONCE.........\n");
        printf("Header Hash: ");
        sha256(header, sizeof(header), fp_new, 0);
        printf("Nonce Hash: ");
        sha256(nonce + 161, sizeof(nonce), fp_new, 0);

        /** Concatenate and hash + nonce and hash the concatenation */
        printf("\nVALIDATING BLOCK HASH.........");
        fgets((char *)concat, 226, fp_newer); 
        concat[strcspn((char *)concat, "\n")] = 0;
        printf("\n");
        printf("Concat Hash: ");
        sha256(concat, sizeof(concat), fp_new, 1);
        counter++;
    }
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    
    /** Record execution time */
    printf("\n\nTime elapsed is: %f\n", time_taken);

    if (counter == 16222) {
      printf("BLOCKCHAIN VALIDATED!");
    }

    fclose(fp);
    fclose(fp_new);
    fclose(fp_newer);
} 